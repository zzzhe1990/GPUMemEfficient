
#include <hip/hip_runtime.h>
#include<iostream>
#include<cstdlib>
#include<fstream>
#include<string>
#include<sys/time.h>

typedef unsigned long long int UINT;
const int MAXTHREADSPERBLOCK = 512;

using namespace std;


__global__ void GPU(const int tilesize, const int paddsize, const int maxThreads, 
			int *dev_table, const int rowsize, const int maxlevel, int tileX, int lenY, int *dev_arr1, int *dev_arr2){
	//This code has to ensure n2 size is the multiple of 128. And n2 is no smaller than n1, where n2 is row array size, n1 is colum array size
	//on K40, tile size is max to 48K, which is 128*96; on pascal and volta, tile size is max to 64K which is 128*128
	//For running on K40, we reserve the shared memory space for a 64*64 tile. Because of the dependency, the actual shared memory size is 96 * 96. 96 is picked for ensuring memory coalscing.
//	__shared__ int table[9216];

	int thread = threadIdx.x;
	int curjobs = 0;
	int curlevel = 1;
	int startIdx, startx, starty;
	int tableX = tileX + paddsize;
	int x,y;
	
	while (curlevel <= maxlevel){
		if (curlevel <= lenY){
			curjobs++;
//			printf("curlevel: %d, lenY: %d, curjobs: %d, thread: %d\n", curlevel, lenY, curjobs, thread);
		}

		startx = paddsize + curlevel - 1;
		starty = paddsize;

		if (curlevel > tileX){
			curjobs--;	
			startx = tableX -1;
			starty = paddsize + curlevel - tileX;

//			printf("curlevel: %d, curjobs: %d, thread: %d\n", curlevel,  curjobs, thread);
		}
	
		if (thread < curjobs){
			startx -= thread;
			starty += thread;
			startIdx = startx + starty * rowsize;
			dev_table[startIdx] = max(dev_table[startIdx - 1], dev_table[startIdx - rowsize]);
			x = startx - paddsize;
			y = starty - paddsize;
			if (dev_arr1[x] == dev_arr2[y])
				dev_table[startIdx] = dev_table[startIdx - rowsize - 1] + 1;				
		}

		curlevel++;
		
	
		__syncthreads();	
	}
	
}

void checkGPUError(hipError_t err){
	if (hipSuccess != err){
		printf("CUDA error in file %s, in line %i: %s\n", __FILE__, __LINE__, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

int LCS(int n1, int n2, int *arr1, int *arr2){
	int lcslength;
	int paddsize = 1;
	int tileX = 256;
	int tileY = 256;
	int rowsize = paddsize + n2;
	int colsize = paddsize + n1;

	int *dev_table, *dev_arr1, *dev_arr2;

	int *table;

	table = new int[colsize * rowsize];

	size_t freeMem, totalMem;

	hipMemGetInfo(&freeMem, &totalMem);
	int tablesize = colsize * rowsize;
	cout << "current GPU memory info FREE: " << freeMem << " Bytes, Total: " << totalMem << " Bytes.";
	cout << "colsize: " << colsize << ", rowsize: " << rowsize << ", allocates: " << tablesize * sizeof(int)<< " Bytes." << endl;
	hipError_t err = hipMalloc(&dev_table, tablesize * sizeof(int));
	checkGPUError(err);
	
	hipMalloc(&dev_arr1, n1*sizeof(int));
	hipMalloc(&dev_arr2, n2*sizeof(int));

	hipMemset(dev_table, 0, tablesize * sizeof(int));
	hipMemcpy(dev_arr1, arr1, n1*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_arr2, arr2, n2*sizeof(int), hipMemcpyHostToDevice);

	int maxthreads;
	int maxlevel;
	int threadPerBlock = tileY + paddsize;
	int blockPerGrid = (threadPerBlock + MAXTHREADSPERBLOCK -1)/MAXTHREADSPERBLOCK;
	int numStream = 32;

	hipDeviceSetCacheConfig(hipFuncCachePreferShared);

	int xseg = (n1+tileX-1) / tileX;
	int yseg = (n2+tileY-1) / tileY;
	int maxSegThreads = min(xseg, yseg);		//max number of segs at either of the seg levels.
	int maxSegLevel = xseg + yseg - 1;
	int curSegLevel = 1;
//	int curSegJobs = 1;
	int startSegX, startSegY;	

	hipStream_t stream[numStream];
	for (int s=0; s<numStream; s++)
		hipStreamCreate(&stream[s]);

	while(curSegLevel <= maxSegLevel){
		int segIdx = 0;
		if (curSegLevel <= xseg){
			//curSegJobs = curSegLevel;
			startSegX = curSegLevel - 1;
			startSegY = 0;
		}	
		else{
			//startSegIdx = xseg - 1 + xseg * (curSegLevel - xseg);
			//curSegJobs = 2 * xseg - curSegLevel;
			startSegX = xseg - 1;
			startSegY = curSegLevel - xseg;
		}
		
//		cout << "curSegLevel: " << curSegLevel << ", maxSegLevel: " << maxSegLevel << endl;	
		
		while ( startSegX >= 0 && startSegY <= yseg - 1){
			//suppose n2 is the row size and the longer array
			//int i = paddsize + startSegX * tileX;
			//int j = paddsize + startSegY * tileY;
			int i = startSegX * tileX;
			int j = startSegY * tileY;
			int startSegAdd = j * rowsize + i;
			int s = segIdx % numStream;
			//resY is used to determine the rest size of Y. This is used to check if the rest size of Y is smaller than tileY.
			int resY = n1 - startSegY * tileY;
			int lenY = min(resY, tileY);
			maxlevel = tileX + lenY - 1;
			maxthreads = min(tileX, lenY);
			int tilesize = (tileX+paddsize) * (lenY+paddsize);

			GPU<<<blockPerGrid, threadPerBlock, 0, stream[s]>>>(tilesize, paddsize, maxthreads, &dev_table[startSegAdd], rowsize, 
										maxlevel, tileX, lenY, &dev_arr1[i], &dev_arr2[j]);
		
//			cout << "startSegX: " << startSegX << ", startSegY: " << startSegY << ", segIdx: " << segIdx << endl;
			startSegX--;
			startSegY++;
			segIdx++;
		}
		//this synchronization is might removable
		hipDeviceSynchronize();

		curSegLevel++;
	}
	
	hipMemcpy(&lcslength, &dev_table[tablesize-1], sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(table, dev_table, (n1+paddsize)*rowsize*sizeof(int), hipMemcpyDeviceToHost);
/*
	//display table
	cout << "full table: " << endl;
	for (int i=0; i<n1+paddsize; i++){
		for (int j=0; j<n2+paddsize; j++){
			cout << table[i * rowsize + j] << " ";
		}
		cout << endl;
	}
*/	

	for (int s=0; s<numStream; s++)
		hipStreamDestroy(stream[s]);
	
	hipFree(dev_arr1);
	hipFree(dev_arr2);
	hipFree(dev_table);

	delete[] table;

	return lcslength;
}

